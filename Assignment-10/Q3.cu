#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
using std namespace;



__global__ void prefixSum(int *v, int *v_r) {

  __shared__ int partial_sum[SHMEM_SIZE];
  // Calculate thread ID
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
 
  partial_sum[threadIdx.x] = v[tid];
  __syncthreads();
  
  for (int s = 1; s < blockDim.x; s *= 2) {
   
    if (threadIdx.x % (2 * s) == 0) {
      partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
    }
    __syncthreads();
  }
  
}


int main() {
  // Vector size
  int N = 1 << 16;
  size_t bytes = N * sizeof(int);
  // Host data
  vector<int> h_v(N);
  vector<int> h_v_r(N);
  // Initialize the input data
  generate(begin(h_v), end(h_v), []() { return rand() % 10; });
  // Allocate device memory
  int *d_v, *d_v_r;
  hipMalloc(&d_v, bytes);
  hipMalloc(&d_v_r, bytes);
  // Copy to device
  hipMemcpy(d_v, h_v.data(), bytes, hipMemcpyHostToDevice);
  // TB Size
  const int TB_SIZE = 256;
  // Grid Size (No padding)
  int GRID_SIZE = N / TB_SIZE;
  // Call kernels
  prefixSum<<<1, TB_SIZE>>>(d_v_r, d_v_r);
  // Copy to host;
  hipMemcpy(h_v_r.data(), d_v_r, bytes, hipMemcpyDeviceToHost);
  // Print the result
  assert(h_v_r[0] == std::accumulate(begin(h_v), end(h_v), 0));
  cout << "COMPLETED SUCCESSFULLY\n";
  return 0;
}
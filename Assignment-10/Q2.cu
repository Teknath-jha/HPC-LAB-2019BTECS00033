
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define TILE_WIDTH 2



/*matrix multiplication kernels*/

// shared
__global__ void
MatrixMulSh( float *Md , float *Nd , float *Pd , const int WIDTH )
{
	
	//Taking shared array to break the MAtrix in Tile widht and fatch them in that array per ele
	
	__shared__ float Mds [TILE_WIDTH][TILE_WIDTH] ;
	
	__shared__ float Nds [TILE_WIDTH][TILE_WIDTH] ;
	
	// calculate thread id
	unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x ;
	unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y ;
	
	for (int m = 0 ; m<WIDTH/TILE_WIDTH ; m++ ) // m indicate number of phase
	{
		Mds[threadIdx.y][threadIdx.x] =  Md[row*WIDTH + (m*TILE_WIDTH + threadIdx.x)]  ;
		Nds[threadIdx.y][threadIdx.x] =  Nd[ ( m*TILE_WIDTH + threadIdx.y) * WIDTH + col] ;
		__syncthreads() ; // for syncronizeing the threads
		
		// Do for tile
		for ( int k = 0; k<TILE_WIDTH ; k++ )
			Pd[row*WIDTH + col]+= Mds[threadIdx.x][k] * Nds[k][threadIdx.y] ;
		__syncthreads() ; // for syncronizeing the threads
		
	}
}

// main routine
int main ()
{
	const int WIDTH = 500;
	float array1_h[WIDTH][WIDTH] ,array2_h[WIDTH][WIDTH], M_result_array_h[WIDTH][WIDTH]  ;
	float *array1_d , *array2_d ,*result_array_d  ,*M_result_array_d ; // device array
	int i , j ;
	//input in host array
	for ( i = 0 ; i<WIDTH ; i++ )
	{
		for (j = 0 ; j<WIDTH ; j++ )
		{
			array1_h[i][j] = (i + 2*j) %500 ;
			array2_h[i][j] = (i + 3*j) %500 ;
		}
	}
	
	//create device array cudaMalloc ( (void **)&array_name, sizeofmatrixinbytes) ;
	
	hipMalloc((void **) &array1_d , WIDTH*WIDTH*sizeof (int) ) ;
	
	hipMalloc((void **) &array2_d , WIDTH*WIDTH*sizeof (int) ) ;
	
	
	
	//copy host array to device array; cudaMemcpy ( dest , source , WIDTH , direction )
	
	hipMemcpy ( array1_d , array1_h , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;
	
	hipMemcpy ( array2_d , array2_h , WIDTH*WIDTH*sizeof (int) , hipMemcpyHostToDevice ) ;
	
	
	
	//allocating memory for resultent device array
	
	hipMalloc((void **) &result_array_d , WIDTH*WIDTH*sizeof (int) ) ;
	
	hipMalloc((void **) &M_result_array_d , WIDTH*WIDTH*sizeof (int) ) ;
	
	
	MatrixMulSh<<<512,32>>> ( array1_d , array2_d ,M_result_array_d , WIDTH) ;
	
	// all gpu function blocked till kernel is working
	//copy back result_array_d to result_array_h
	
	hipMemcpy(M_result_array_h , M_result_array_d , WIDTH*WIDTH*sizeof(int) ,hipMemcpyDeviceToHost) ;
	
    printf("Multiplication Successful using shared Memory");
	
	
}



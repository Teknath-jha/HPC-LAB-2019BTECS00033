#include "hip/hip_runtime.h"
include<stdio.h>
#define row1 20
#define col1 30
#define row2 30
#define col2 20

    __global__ void matproductsharedmemory(int *l, int *m, int *n) {
  int x = blockIdx.x;
  int y = blockIdx.y;
  __shared__ int p[col1];
  int i;
  int k = threadIdx.x;
  n[col2 * y + x] = 0;
  p[k] = l[col1 * y + k] * m[col2 * k + x];
  __syncthreads();
  for (i = 0; i < col1; i++)
    n[col2 * y + x] = n[col2 * y + x] + p[i];
}
int main() {
  int a[row1][col1];
  int b[row2][col2];
  int c[row1][col2];
  int *d, *e, *f;
  int i, j;
  for (i = 0; i < row1; i++) {
    for (j = 0; j < col1; j++) {
      a[i][j] = 2;
    }
  }
  for (i = 0; i < row2; i++) {
    for (j = 0; j < col2; j++) {
      b[i][j] = 3;
    }
  }
  hipMalloc((void **)&d, row1 * col1 * sizeof(int));
  hipMalloc((void **)&e, row2 * col2 * sizeof(int));
  hipMalloc((void **)&f, row1 * col2 * sizeof(int));
  hipMemcpy(d, a, row1 * col1 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(e, b, row2 * col2 * sizeof(int), hipMemcpyHostToDevice);
  dim3 grid(col2, row1);
  matproductsharedmemory<<<grid, col1>>>(d, e, f);
  hipDeviceSynchronize();
  hipMemcpy(c, f, row1 * col2 * sizeof(int), hipMemcpyDeviceToHost);
  for (i = 0; i < row1; i++) {
    for (j = 0; j < col2; j++) {
      if (c[i][j] != 180) {
        printf("False\n");
        return -1;
      }
    }
  }
  hipFree(d);
  hipFree(e);
  hipFree(f);
  printf("True\n");
  return 0;
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#define row1 20
#define col1 30
#define row2 30
#define col2 20


__global__ void matmul(int *l, int *m, int *n) {
  int x = threadIdx.x;
  int y = threadIdx.y;
  int k;
  n[col2 * y + x] = 0;
  for (k = 0; k < col1; k++) {
    n[col2 * y + x] = n[col2 * y + x] + l[col1 * y + k] * m[col2 * k + x];
  }
}


int main() {
  int a[row1][col1];
  int b[row2][col2];
  int c[row1][col2];
  int *d, *e, *f;
  int i, j;
  for (i = 0; i < row1; i++) {
    for (j = 0; j < col1; j++) {
      a[i][j] = 2;
    }
  }
  for (i = 0; i < row2; i++) {
    for (j = 0; j < col2; j++) {
      b[i][j] = 3;
    }
  }
  hipMalloc((void **)&d, row1 * col1 * sizeof(int));
  hipMalloc((void **)&e, row2 * col2 * sizeof(int));
  hipMalloc((void **)&f, row1 * col2 * sizeof(int));


  hipMemcpy(d, a, row1 * col1 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(e, b, row2 * col2 * sizeof(int), hipMemcpyHostToDevice);

  
  dim3 threadBlock(col2, row1);
  matmul<<<1, threadBlock>>>(d, e, f);
  hipDeviceSynchronize();
  hipMemcpy(c, f, row1 * col2 * sizeof(int), hipMemcpyDeviceToHost);
  for (i = 0; i < row1; i++) {
    for (j = 0; j < col2; j++) {
      if (c[i][j] != 180) {
        printf("False\n");
        return -1;
      }
    }
  }
  hipFree(d);
  hipFree(e);
  hipFree(f);
  printf("True\n");
  return 0;
}